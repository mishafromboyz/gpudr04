
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

const int vsize = 512;
const int blocksize = 256;

__global__
void vsum(float *x, float *y, float *z)
{
	//int i = blockIdx.x * blockDim.x + threadIdx.x; @ blockdim, not block_size @
	int i = blockIdx.x * blocksize + threadIdx.x;
	if(i < vsize)
		z[i] = x[i]+y[i]; 
}



int main()
{
	float *A = (float*)malloc(vsize*sizeof(float));
	float *B = (float*)malloc(vsize*sizeof(float));
	float *C = (float*)malloc(vsize*sizeof(float));


	for(int i = 1; i<=vsize; i++)
	{
		A[i]=(float)i;
		B[i]=(float)i;
		//C[i]=(float)i;
	}



	float *dA, *dB, *dC;
	hipMalloc(&dA, vsize * sizeof(float));
	hipMalloc(&dB, vsize * sizeof(float));
	hipMalloc(&dC, vsize * sizeof(float));


	hipMemcpy(dA, A, vsize*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dB, B, vsize*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dC, C, vsize*sizeof(float), hipMemcpyHostToDevice);

	vsum<<<(vsize/blocksize), blocksize>>>(A, B, C); 
	hipDeviceSynchronize();

	
	for (int i = 0; i<vsize; i+=16)
	{
		for (int j = 0; j<32; j++)
			cout << C[j] << " ";
		cout << endl;
	}

	hipFree(dA);
	hipFree(dB);
	hipFree(dC);
	free(A);
	free(B);
	free(C);

	return 0;
}